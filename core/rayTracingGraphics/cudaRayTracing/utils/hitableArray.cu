#include "hip/hip_runtime.h"
#include "hitableArray.h"

#include "operations.h"

namespace cuda::rayTracing {

__host__ __device__ HitableArray::~HitableArray() {
    if (array) {
        delete[] array;
    }
    container_size = 0;
}

__host__ __device__ bool HitableArray::hit(const ray& r, HitCoords& coord) const {
    for(iterator it = begin(); it != end(); it++){
        if ((*it)->hit(r, coord)) {
            coord.obj = *it;
        }
    }
    return coord.obj;
}

__host__ __device__ void HitableArray::add(const Hitable*const* object, size_t size) {
    Pointer* newArray = new Pointer[container_size + size];
    for(size_t i = 0; i < container_size; i++){
        newArray[i] = array[i];
    }
    for(size_t i = 0; i < size; i++){
        newArray[container_size + i].p = object[i];
    }
    if (array) {
        delete[] array;
    }
    array = newArray;
    container_size += size;
}

__host__ __device__ const Hitable*& HitableArray::operator[](uint32_t i) const {
    return array[i].p;
}

__global__ void createKernel(HitableArray* p) {
    p = new (p) HitableArray();
}

void HitableArray::create(HitableArray* dpointer, const HitableArray& host){
    createKernel<<<1,1>>>(dpointer);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
}

__global__ void destroyKernel(HitableArray* p) {
    p->~HitableArray();
}

void HitableArray::destroy(HitableArray* dpointer){
    destroyKernel<<<1,1>>>(dpointer);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
}

}
