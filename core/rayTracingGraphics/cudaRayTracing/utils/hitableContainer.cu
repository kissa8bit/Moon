#include "hip/hip_runtime.h"
#include "hitableContainer.h"

#include "operations.h"
#include "buffer.h"

namespace cuda::rayTracing {

__global__ void addKernel(HitableContainer* container, const Hitable* const* object, size_t size) {
    container->add(object, size);
}

void HitableContainer::add(HitableContainer* dpointer, const std::vector<const Hitable*>& objects) {
    Buffer<const Hitable*> devBuffer(objects.size(), objects.data());
    addKernel<<<1,1>>>(dpointer, devBuffer.get(), devBuffer.getSize());
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
}

__global__ void destroyKernel(HitableContainer* p) {
    p->~HitableContainer();
}

void HitableContainer::destroy(HitableContainer* dpointer){
    destroyKernel<<<1,1>>>(dpointer);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
}

}
